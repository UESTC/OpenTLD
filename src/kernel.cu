#include "hip/hip_runtime.h"
#include "kernel.h"

__global__ void kernel(int size, int *dev_tmp)
{
	*dev_tmp = size + 1;
}

void processWithCuda(TLD *tld){
	TLD *dev_tld = NULL;
	int *dev_tmp = 0;
	int tmp = 0;
	hipSetDevice(0);
	hipMalloc((void**)&dev_tmp, sizeof(int));
	hipMalloc((void**)&dev_tld, sizeof(TLD));
	hipMemcpy(dev_tmp, &tmp, sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_tld, tld, sizeof(TLD), hipMemcpyHostToDevice);
	printf("%d\n", tld->grid.size());
	kernel << <1, 1 >> >(tld->grid.size(), dev_tmp);
	hipMemcpy(&tmp, dev_tmp, sizeof(int), hipMemcpyDeviceToHost);
	printf("%d\n", tmp);
}
